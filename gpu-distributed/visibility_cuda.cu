#include "hip/hip_runtime.h"


// left_offset is the number of pixels from the first from_pixel to the edge of the altitude data on the left


uint16_t* run_visibility_search(
        uint16_t* altitude_data, 
        int altitude_data_size,
        int left_offset,
        dimensions altitude_dim,
        dimensions from_point_dim,
        Point* offsets,
        dimensions block_dim,
        uint16_t* visibility_results) {

    // Needs alitidue data and data dimentions
    // Needs from point dimentions

    // All data will start from the very first thread

    // Allocate memory for Altitude data
    uint16_t* altitude_data_d;
    hipMalloc((void **) &altitude_data_d, altitude_data_size * sizeof(uint16_t));
    hipMemcpy(altitude_data_d, altitude_data, altitude_data_size * sizeof(uint16_t), hipMemcpyHostToDevice);

    // Allocate memory for visibility results for each to point in altitude data (will be same size)
    uint16_t* visibility_results_d;
    hipMalloc((void **) &visibility_results_d, altitude_data_size * sizeof(uint16_t));

    // Allocate memory for from point data (will be same size as from point dimensions)
    uint16_t* from_point_visibility_d;
    hipMalloc((void **) &from_point_visibility_d, from_point_dim.x_width * from_point_dim.y_height * sizeof(uint16_t));

    // Allocate memory for offset values
    Point* offsets_d;
    hipMalloc((void **) &offsets_d, from_point_dim.x_width * from_point_dim.y_height * sizeof(Point));

    

    // Build out block and grid dimensions
    dim3 input_block_dim(block_dim.x_width, block_dim.y_height);
    dim3 grid_dim((int)ceil((float)altitude_dim.x_width / block_dim.x_width), (int)ceil((float)altitude_dim.y_height / block_dim.y_height));
    
    get_visibility_gpu<<<grid_dim, input_block_dim>>>(
        altitude_data_d,
        altitude_data_size,
        left_offset,
        altitude_dim,
        from_point_dim,
        offsets_d,
        visibility_results_d,
        from_point_visibility_d
    );


    
}

__global__ get_visibility_gpu(
        uint16_t* altitude_data,
        int altitude_data_size,
        int left_offset,
        dimensions altitude_dim,
        dimensions from_points_dim,
        Point* offsets,
        uint16_t* visibility_results_d,
        uint16_t* from_point_visibility_d) {


    // Iterate through all the offsets

    // determine if the offset is within the bounds of the altitude data

    // If it is, check if the point is visible from the from point
        // If it is, increment the visibility result for this point in the from point visibility data as well as in the to point data
    // If it is not, do nothing

    Point from_section_xy = {blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y}
    Point a_data_xy = {from_section_x + left_offset, blockIdx.y * blockDim.y + threadIdx.y};

    // Check if the thread is within the bounds of the from_point dimensions, exits if it is
    if (from_section_xy.x_width > from_points_dim.x_width || from_section_xy.y_height > from_points_dim.y_height) {
        return;
    }

    for (int i = 0; i < 20200; i++) {
        
        Point current_offset = offsets[i];
        // Get the xy coordinates of the offset within the altitude data
        Point a_data_to_xy = {a_data_xy.x_width + current_offset.x_width, a_data_xy.y_height + current_offset.y_height};
        // Check if the offset is within the bounds of the altitude data
        if (a_data_to_xy.x_width < 0 || a_data_to_xy.x_width >= altitude_dim.x_width || a_data_to_xy.y_height < 0 || a_data_to_xy.y_height >= altitude_dim.y_height) {
            continue;
        }

        // Get the visibility between the main point and the offset point
        float slope = visibility_line_slope(
            altitude_data[a_data_xy.y_height * altitude_dim.x_width + a_data_xy.x_width],
            altitude_data[a_data_to_xy.y_height * altitude_dim.x_width + a_data_to_xy.x_width],
            a_data_xy.x_width,
            a_data_xy.y_height,
            a_data_to_xy.x_width,
            a_data_to_xy.y_height
        );
        bool visibile = visibility_path(altitude_data, slope, a_data_xy.x_width, a_data_xy.y_height, a_data_to_xy.x_width, a_data_to_xy.y_height, altitude_dim.x_width);

        // If the point is visible, increment the visibility result for this point in the from point visibility data as well as in the to point data
        if (visibile) {
            // Increment the visibility result for this point in the from point visibility data
            from_point_visibility_d[from_section_xy.y_height * from_points_dim.x_width + from_section_xy.x_width] += 1;
            // Increment the visibility result for this point in the to point data
            visibility_results_d[a_data_to_xy.y_height * altitude_dim.x_width + a_data_to_xy.x_width] += 1;
        }
    }


}

__device__ bool visibility_path(uint16_t* altitude_data, float slope, int x1, int y1, int x2, int y2, int a_data_width)
{

    // std::pair<int, int> point;
    short altitude = altitude_data[y1 * a_data_width + x1];

	// Compute the differences between start and end points
	int dx = x2 - x1;
	int dy = y2 - y1;

	// Absolute values of the change in x and y
	const int abs_dx = abs(dx);
	const int abs_dy = abs(dy);

	// Initial point
	int x = x1;
	int y = y1;

	// Proceed based on the absolute differences to support all octants
	if (abs_dx > abs_dy)
	{
		// If the line is moving to the left, set dx accordingly
		int dx_update;
		if (dx > 0)
		{
			dx_update = 1;
		}
		else
		{
			dx_update = -1;
		}

		// Calculate the initial decision parameter
		int p = 2 * abs_dy - abs_dx;

		// Draw the line for the x-major case
		for (int i = 0; i <= abs_dx; i++)
		{
            Point point = {x, y};
            if (x != x1 && y != y1 && x != x2 && y != y2) {
                if ((altitude_data[point.y_height * a_data_width + point.x_width]) < (altitude + slope)) {
                    altitude = altitude + slope;
                }
                else {
                    return false;
                }
            }

			// Threshold for deciding whether or not to update y
			if (p < 0)
			{
				p = p + 2 * abs_dy;
			}
			else
			{
				// Update y
				if (dy >= 0)
				{
					y += 1;
				}
				else
				{
					y += -1;
				}

				p = p + 2 * abs_dy - 2 * abs_dx;
			}

			// Always update x
			x += dx_update;
		}
	}
	else
	{
		// If the line is moving downwards, set dy accordingly
		int dy_update;
		if (dy > 0)
		{
			dy_update = 1;
		}
		else
		{
			dy_update = -1;
		}

		// Calculate the initial decision parameter
		int p = 2 * abs_dx - abs_dy;

		// Draw the line for the y-major case
		for (int i = 0; i <= abs_dy; i++)
		{
            Point point = {x, y};
            if (x != x1 && y != y1 && x != x2 && y != y2) {
                if ((altitude_data[point.y_height * a_data_width + point.x_width]) < (altitude + slope)) {
                    altitude = altitude + slope;
                }
                else {
                    return false;
                }
            }

			// Threshold for deciding whether or not to update x
			if (p < 0)
			{
				p = p + 2 * abs_dx;
			}
			else
			{
				// Update x
				if (dx >= 0)
				{
					x += 1;
				}
				else
				{
					x += -1;
				}

				p = p + 2 * abs_dx - 2 * abs_dy;
			}

			// Always update y
			y += dy_update;
		}
	}

    return true;
}

__device__ float visibility_line_slope(short starting_altitude, short ending_altitude, short x1, short y1, short x2, short y2) {
    return (ending_altitude - starting_altitude) / sqrt(pow(x2 - x1, 2) + pow(y2 - y1, 2));
}

Point* pixelList_offset() {

    // Create a place to store pixel values
    Point* pixels = (Point*)malloc(sizeof(Point) * 20200);

    // Set the bounds of the relevant pixel box
    int starting_x = -100;
    int stopping_x = 100;
    int starting_y = 0;
    int stopping_y = 100;

    int place = 0;

    for (int y = starting_y; y <= stopping_y; y++) {
        for (int x = starting_x; x <= stopping_x; x++) {
            if (y == starting_y && x <= 0) continue;
            pixels[place] = { x, y };
            place++;
        }
    }

    // This code assumes x values go across and y values up and down

    return pixels;
}










