#include "hip/hip_runtime.h"
#include "visibility_cuda.hpp"

__global__ void get_visibility_gpu(uint16_t* altitude_data, dimensions altitude_dim, int left_offset, dimensions from_points_dim, Point* offsets, int* visibility_results_d, int* from_point_visibility_d);
__device__ bool visibility_path(uint16_t* altitude_data, float slope, int x1, int y1, int x2, int y2, int a_data_width);
__device__ float visibility_line_slope(short starting_altitude, short ending_altitude, short x1, short y1, short x2, short y2);


int* run_visibility_search(
        uint16_t* altitude_data, 
        dimensions altitude_dim,
        int left_offset,
        dimensions from_point_dim,
        Point* offsets,
        dimensions block_dim,
        int* visibility_results,
        int rank) {


    // Allocate memory for Altitude data
    uint16_t* altitude_data_d;
    int altitude_data_size = altitude_dim.x_width * altitude_dim.y_height;
    hipMalloc((void **) &altitude_data_d, altitude_data_size * sizeof(uint16_t));
    hipMemcpy(altitude_data_d, altitude_data, altitude_data_size * sizeof(uint16_t), hipMemcpyHostToDevice);

    // Allocate memory for visibility results for each to-point in altitude data (will be same size)
    int* visibility_results_d;
    hipMalloc((void **) &visibility_results_d, altitude_data_size * sizeof(int));

    // Allocate memory for from point data (will be same size as from point dimensions)
    int* from_point_visibility_d;
    hipMalloc((void **) &from_point_visibility_d, from_point_dim.x_width * from_point_dim.y_height * sizeof(int));

    // Allocate memory for offset values
    Point* offsets_d;
    hipMalloc((void **) &offsets_d, 19999 * sizeof(Point));
    hipMemcpy(offsets_d, offsets, 19999 * sizeof(Point), hipMemcpyHostToDevice);

    // Build out the grid and block dimensions
    dim3 input_block_dim(block_dim.x_width, block_dim.y_height);
    dim3 grid_dim((int)ceil((float)from_point_dim.x_width / block_dim.x_width), (int)ceil((float)from_point_dim.y_height / block_dim.y_height));

    fflush(stdout);

    // Figure out which CUDA device to use based on the rank
    int device_count;
    hipGetDeviceCount(&device_count);
    // printf("Device Count: %d\n", device_count);
    hipSetDevice(rank % device_count);

    get_visibility_gpu<<<grid_dim, input_block_dim>>>(
        altitude_data_d,
        altitude_dim,
        left_offset,
        from_point_dim,
        offsets_d,
        visibility_results_d,
        from_point_visibility_d
    );

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
    }
    hipDeviceSynchronize();

    // Copy the visibility results back to the host
    hipMemcpy(visibility_results, visibility_results_d, altitude_data_size * sizeof(int), hipMemcpyDeviceToHost);

    // Copy the from point visibility results back to the host
    int* from_point_visibility = (int*)malloc(from_point_dim.x_width * from_point_dim.y_height * sizeof(int));
    hipMemcpy(from_point_visibility, from_point_visibility_d, from_point_dim.x_width * from_point_dim.y_height * sizeof(int), hipMemcpyDeviceToHost);
    
    // Combine the from_point data and to_point data
    for (int x = 0; x < from_point_dim.x_width; x++) {
        for (int y = 0; y < from_point_dim.y_height; y++) {
            visibility_results[y * altitude_dim.x_width + x + left_offset] += from_point_visibility[y * from_point_dim.x_width + x];
        }
    }

    return 0;
}

__global__ void get_visibility_gpu(
        uint16_t* altitude_data,
        dimensions altitude_dim,
        int left_offset,
        dimensions from_points_dim,
        Point* offsets,
        int* visibility_results_d,
        int* from_point_visibility_d) {

    // Get XY position relative to the altitude data as well as the from_point section
    Point from_section_xy = {(int)(blockIdx.x * blockDim.x + threadIdx.x), (int)(blockIdx.y * blockDim.y + threadIdx.y)};
    Point a_data_xy = {(int)(blockIdx.x * blockDim.x + threadIdx.x + left_offset), (int)(blockIdx.y * blockDim.y + threadIdx.y)};

    // Check if the thread is within the bounds of the from_point dimensions, exits if it is
    if (from_section_xy.x >= from_points_dim.x_width || from_section_xy.y >= from_points_dim.y_height) {
        return;
    }

    for (int i = 0; i < 19999; i++) {
        
        Point current_offset = offsets[i];

        // Get the xy coordinates of the offset within the altitude data
        Point a_data_to_xy = {a_data_xy.x + current_offset.x, a_data_xy.y + current_offset.y};

        // Check if the offset is within the bounds of the altitude data
        if (a_data_to_xy.x < 0 || a_data_to_xy.x >= altitude_dim.x_width || a_data_to_xy.y < 0 || a_data_to_xy.y >= altitude_dim.y_height) {
            continue;
        }

        // Get the visibility between the main point and the offset point
        float slope = visibility_line_slope(
            altitude_data[a_data_xy.y * altitude_dim.x_width + a_data_xy.x],
            altitude_data[a_data_to_xy.y * altitude_dim.x_width + a_data_to_xy.x],
            a_data_xy.x,
            a_data_xy.y,
            a_data_to_xy.x,
            a_data_to_xy.y
        );

        bool visible = visibility_path(altitude_data, slope, a_data_xy.x, a_data_xy.y, a_data_to_xy.x, a_data_to_xy.y, altitude_dim.x_width);

        // If the point is visible, increment the visibility result for this point in the from point visibility data as well as in the to point data
        if (visible) {

            // Increment the visibility result for this point in the from point visibility data
            atomicAdd(&from_point_visibility_d[from_section_xy.y * from_points_dim.x_width + from_section_xy.x], 1);

            // Increment the visibility result for this point in the to point data
            atomicAdd(&visibility_results_d[a_data_to_xy.y * altitude_dim.x_width + a_data_to_xy.x], 1);
        }
    }
}

__device__ bool visibility_path(uint16_t* altitude_data, float slope, int x1, int y1, int x2, int y2, int a_data_width)
{

    // std::pair<int, int> point;
    short altitude = altitude_data[y1 * a_data_width + x1];

	// Compute the differences between start and end points
	int dx = x2 - x1;
	int dy = y2 - y1;

	// Absolute values of the change in x and y
	const int abs_dx = abs(dx);
	const int abs_dy = abs(dy);

	// Initial point
	int x = x1;
	int y = y1;

	// Proceed based on the absolute differences to support all octants
	if (abs_dx > abs_dy)
	{
		// If the line is moving to the left, set dx accordingly
		int dx_update;
		if (dx > 0)
		{
			dx_update = 1;
		}
		else
		{
			dx_update = -1;
		}

		// Calculate the initial decision parameter
		int p = 2 * abs_dy - abs_dx;

		// Draw the line for the x-major case
		for (int i = 0; i <= abs_dx; i++)
		{
            Point point = {x, y};
            if (x != x1 && y != y1 && x != x2 && y != y2) {
                if ((altitude_data[point.y * a_data_width + point.x]) < (altitude + slope)) {
                    altitude = altitude + slope;
                }
                else {
                    return false;
                }
            }

			// Threshold for deciding whether or not to update y
			if (p < 0)
			{
				p = p + 2 * abs_dy;
			}
			else
			{
				// Update y
				if (dy >= 0)
				{
					y += 1;
				}
				else
				{
					y += -1;
				}

				p = p + 2 * abs_dy - 2 * abs_dx;
			}

			// Always update x
			x += dx_update;
		}
	}
	else
	{
		// If the line is moving downwards, set dy accordingly
		int dy_update;
		if (dy > 0)
		{
			dy_update = 1;
		}
		else
		{
			dy_update = -1;
		}

		// Calculate the initial decision parameter
		int p = 2 * abs_dx - abs_dy;

		// Draw the line for the y-major case
		for (int i = 0; i <= abs_dy; i++)
		{
            Point point = {x, y};
            if (x != x1 && y != y1 && x != x2 && y != y2) {
                if ((altitude_data[point.y * a_data_width + point.x]) < (altitude + slope)) {
                    altitude = altitude + slope;
                }
                else {
                    return false;
                }
            }

			// Threshold for deciding whether or not to update x
			if (p < 0)
			{
				p = p + 2 * abs_dx;
			}
			else
			{
				// Update x
				if (dx >= 0)
				{
					x += 1;
				}
				else
				{
					x += -1;
				}

				p = p + 2 * abs_dx - 2 * abs_dy;
			}

			// Always update y
			y += dy_update;
		}
	}

    return true;
}

__device__ float visibility_line_slope(short starting_altitude, short ending_altitude, short x1, short y1, short x2, short y2) {
    return (ending_altitude - starting_altitude) / sqrt(pow(x2 - x1, 2) + pow(y2 - y1, 2));
}

Point* pixelList_offset() {

    // Create a place to store pixel values
    Point* pixels = (Point*)malloc(sizeof(Point) * 19999);

    // Set the bounds of the relevant pixel box
    int starting_x = -100;
    int stopping_x = 100;
    int starting_y = 0;
    int stopping_y = 99;

    int place = 0;

    for (int y = starting_y; y <= stopping_y; y++) {
        for (int x = starting_x; x <= stopping_x; x++) {
            if (y == starting_y && x <= 0) continue;
            pixels[place] = { x, y };
            place++;
        }
    }

    return pixels;
}










